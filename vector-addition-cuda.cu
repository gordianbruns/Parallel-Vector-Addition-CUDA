
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <getopt.h>


__global__ void add_vector(int *vOne, int *vTwo, int *vResult, int N) {
	int i;

	i = blockDim.x * blockIdx.x + threadIdx.x;

	while (i < N) {
		vResult[i] = vOne[i] + vTwo[i];
		i += blockDim.x;
	}
}
int main(int argc, char* argv[]) {
	int numThreadBlocks, numThreadsPerBlock, size, N, i;
	int *vOne, *vTwo, *vResult;
	int *gpu_vOne, *gpu_vTwo, *gpu_vResult;
	hipError_t status = (hipError_t)0;
	int opt = 0, debug = 0;
	char err_msg[128] = "usage: ./vector-addition-cuda -b <number thread blocks> -t <number threads per block> -n <items>\n";
	

	while ((opt = getopt(argc, argv, "d:b:n:t:")) != -1) {
	    switch(opt) {
		case 'd':
		    debug = 1;
		    break;
		case 'b':
		    numThreadBlocks = atoi(optarg);
		    break;
		case 't':
		    numThreadsPerBlock = atoi(optarg);
		    break;
		case 'n':
		    N = atoi(optarg);
		    break;
		default:
		    printf("Use -d for debugging\n");
		    fprintf(stderr, err_msg);
		    exit(-1);
	    }
	}

	size = N * sizeof(int);

	vOne = (int*) malloc(size);
	vTwo = (int*) malloc(size);
	vResult = (int*) malloc(size);

	if ((vOne == NULL) || (vTwo == NULL) || (vResult == NULL)) {
		perror("initial malloc() of mOne, mTwo, and/or mResult failed");
		exit(-1);
	}

	if (numThreadBlocks <= 0 || numThreadsPerBlock <= 0 || N <= 0) {
		fprintf(stderr, err_msg);
		exit(-1);
	}

	if (debug) {
		printf("numThreadBlocks: %d, numThreadsPerBlock: %d, # items: %d\n", numThreadBlocks, numThreadsPerBlock, N);
	}

	for (i = 0; i < N; i++) {
		vOne[i] = 3333;
		vTwo[i] = 7777;
		vResult[i] = 0;
	}

	if ((status = hipMalloc ((void**) &gpu_vOne, size)) != hipSuccess) {
		printf("cudaMalloc() FAILED (Block), status = %d (%s)\n", status, hipGetErrorString(status));
		exit(1);
	}

	if ((status = hipMalloc ((void**) &gpu_vTwo, size)) != hipSuccess) {
		printf("cudaMalloc() FAILED (Thread), status = %d (%s)\n", status, hipGetErrorString(status));
		exit(1);
	}

	if ((status = hipMalloc ((void**) &gpu_vResult, size)) != hipSuccess) {
		printf("cudaMalloc() FAILED (GThread), status = %d (%s)\n", status, hipGetErrorString(status));
		exit(1);
	}
	
	hipMemcpy(gpu_vOne, vOne, size, hipMemcpyHostToDevice);
	hipMemcpy(gpu_vTwo, vTwo, size, hipMemcpyHostToDevice);

	add_vector <<<numThreadBlocks, numThreadsPerBlock>>>
	  (gpu_vOne, gpu_vTwo, gpu_vResult, N);

	
	hipMemcpy(vResult, gpu_vResult, size, hipMemcpyDeviceToHost);

	#ifdef DISPLAY
	for (i = 0; i < 2; i++) {
		printf("vResult[%d] = %d\n", i, vResult[i]);
	}
	#endif

	free(vOne);
	free(vTwo);
	free(vResult);
	hipFree(gpu_vOne);
	hipFree(gpu_vTwo);
	hipFree(gpu_vResult);

	exit(0);
}
